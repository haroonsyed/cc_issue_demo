#include "hip/hip_runtime.h"
#include <unordered_map>

#include "./cuda_kernels.cuh"

bool init_pool = false;
size_t mat_generated_count(0);
std::unordered_map<size_t, float*> mat_map;

// Error checking macro: https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = false) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void cuda_synchronize() {
    hipDeviceSynchronize();
}

/////////////////////
/// Matrix Setup API
/////////////////////
void init_min_pool_size() {
    int device;
    hipGetDevice(&device);
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, device);
    size_t threshold = sizeof(float) * 2048 * 2048;  // Around 68 Mb reserved
    hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold);
    init_pool = false;
}
size_t register_matrix_buffer(float* gpu_buffer) {
    if (init_pool) {
        init_min_pool_size();
    }

    // Register with the map for retrieval later
    mat_map[mat_generated_count] = gpu_buffer;
    return mat_generated_count++;  // Fine if this overflows
}

size_t register_matrix(size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));

    return register_matrix_buffer(gpu_buffer);
}

size_t register_matrix(float* data, size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));
    gpuErrchk(hipMemcpy(gpu_buffer, data, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    // Potentially nasty bug by acting like you copied data when you havent finished if using cudaMemCpyAsync...
    return register_matrix_buffer(gpu_buffer);
}

void unregister_matrix(size_t mat_id) {
    gpuErrchk(hipFreeAsync(mat_map[mat_id], 0));
    mat_map.erase(mat_id);
}

void get_matrix_data(size_t mat_id, int rows, int cols, float* data_buffer) {
    float* gpu_buffer = mat_map[mat_id];
    gpuErrchk(hipMemcpy(data_buffer, gpu_buffer, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
__global__ void matrix_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    // Go by col row instead of row col. Enabled memory coalescing
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < out_rows && col < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        float weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * row + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + col;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = row * out_cols + col;
        out_buffer[output_index] = weighted_sum;
    }
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 16;
    const int THREADS_PER_BLOCK_Y = 16;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    matrix_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}